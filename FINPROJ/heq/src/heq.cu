#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
 
#include "config.h"

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}

__global__ void calc_hist(unsigned char *input_image, 
                       unsigned int *output_hist){

	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
	  
	int location = 	y*(gridDim.x*TILE_SIZE)+x;
    int val = input_image[location];
    atomicAdd(&(output_hist[val]), 1);

}
                
__global__ void update_px(unsigned char *input_image, 
                       unsigned int *input_lut,
                       unsigned char *output_image){

	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
	  
	int location = 	y*(gridDim.x*TILE_SIZE)+x;
    //output_image = input_lut[input_image[location]];
}

__global__ void warmup(unsigned char *input, 
                       unsigned char *output){

	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
	  
	int location = 	y*(gridDim.x*TILE_SIZE)+x;
	
    output[location] = 0;

}


void gpu_function(unsigned char *data,  
                  unsigned int height, 
                  unsigned int width){
    
    unsigned char *input_gpu;
    unsigned char *output_gpu;

	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	
	int size = XSize*YSize;

    ///////
    // sources
    //      https://hackernoon.com/histogram-equalization-in-python-from-scratch-ebb9c8aa3f23
    //////

    ////////////////////////
    // calculate histogram of image -> write a kernal for this:
    /////////////////////////

    // init hist
    unsigned int hist[256] = {0};

	// Allocate arrays in GPU memory
    // input: image
	//checkCuda(hipMalloc((void**)&input_image, size*sizeof(unsigned char)));
    // output / input: histogram(out) / lut(in)
	//checkCuda(hipMalloc((void**)&output_hist_lut, 256*sizeof(unsigned int)));
    // output: image
	//checkCuda(hipMalloc((void**)&output_image, size*sizeof(unsigned char)));
	
    // zero out histogram and image
    //checkCuda(hipMemset(output_hist_lut, 0, 256*sizeof(unsigned int)));
    //checkCuda(hipMemset(output_image, 0, size*sizeof(unsigned char)));
	
    // copy image to GPU
    //checkCuda(hipMemcpy(input_image, 
    //    data, 
    //    size*sizeof(unsigned char), 
    //    hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());

    // Execute algorithm

    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

	// Kernel Call
	#ifdef CUDA_TIMING
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
        
        // calculate histogram on gpu
        //calc_hist<<<dimGrid, dimBlock>>>(input_image, output_hist_lut);
        for(int ii = 0; ii < size; ++ii) {
            hist[data[ii]] += 1;
        }
        

        // retrieve hist from gpu
        //checkCuda(hipMemcpy(hist, 
        //        output_hist, 
        //        256*sizeof(unsigned int), 
        //        hipMemcpyDeviceToHost));

        ///////////////
        // LUT is a prefix sum! -> bad target for parrallel (unless implement scan)
        // create LUT:
        //      LUT[0] = histogram[0]
        //      LUT[i] = LUT[i-1] + histogram[i]
        //      normalize LUT to 0-255
        //          - sub all by LUT[0], 
        //////////////
        unsigned int lut[256];
        lut[0] = 0;

        for (int ii = 1; ii < 256; ++ii) {
            lut[ii] = (lut[ii-1] + hist[ii]);
        }

        for (int ii = 1; ii < 256; ++ii) {
            lut[ii] = ((1.0f * lut[ii]) / lut[255]) * 255;
        }

        
        
        // write lut to gpu
        //checkCuda(hipMemcpy(output_hist_lut, 
        //        lut, 
         //       256*sizeof(unsigned int), 
          //      hipMemcpyHostToDevice));

        ////////////
        // use LUT to compute new value -> write a kernal for this:
        // output[location] = lut[input[location]]
        ///////////
        //calc_hist<<<dimGrid, dimBlock>>>(input_image, output_hist_lut, output_image);
        for(int ii = 0; ii < size; ++ii) {
            data[ii] = lut[data[ii]];
        }
            
        // From here on, no need to change anything
        checkCuda(hipPeekAtLastError());                                     
        checkCuda(hipDeviceSynchronize());
	
	#ifdef CUDA_TIMING
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif

    // Retrieve results from the GPU
    //checkCuda(hipMemcpy(data,
    //            output_gpu, 
    //            size*sizeof(unsigned char), 
    //            hipMemcpyDeviceToHost));
        
    // Free resources and end the program
	//checkCuda(hipFree(output_image));
	//checkCuda(hipFree(output_hist_lut));
	//checkCuda(hipFree(input_image));

}

void gpu_warmup(unsigned char *data, 
                unsigned int height, 
                unsigned int width){
    
    unsigned char *input_gpu;
    unsigned char *output_gpu;
     
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	
	// Both are the same size (CPU/GPU).
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
	
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
            
    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu, 
        data, 
        size*sizeof(char), 
        hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());
        
    // Execute algorithm
        
	dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    
    warmup<<<dimGrid, dimBlock>>>(input_gpu, 
                                  output_gpu);
                                         
    checkCuda(hipDeviceSynchronize());
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));
                        
    // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));

}

