#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "config.h"

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     
  
unsigned char *input_gpu;
unsigned char *output_gpu;

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}

__global__ void kernel(unsigned char *input, 
                       unsigned char *output,
                       unsigned int size_x,
                       unsigned int size_y,
                       int threshold){

	const int xt_start = max(blockIdx.x*TILE_SIZE, 1);
	const int yt_start = max(blockIdx.y*TILE_SIZE, 1);
	
	const int xt_end = min(xt_start + TILE_SIZE - 1, size_x - 2);
	const int yt_end = min(yt_start + TILE_SIZE - 1, size_y - 2);
    
	const int x = min(xt_start + threadIdx.x, xt_end);
	const int y = min(yt_start + threadIdx.y, yt_end);
    const int dataW = TILE_SIZE*gridDim.x;
	const int location = dataW*y + x;

	const int var1 = input[ dataW * (y-1) + x+1 ] - input[ dataW * (y+1) + x-1 ];
    const int var2 = input[ dataW * (y+1) + x+1 ] - input[ dataW * (y-1) + x-1 ];
	const int var3 = input[ dataW * (y)   + x+1 ] - input[ dataW * (y)   + x-1 ];
	const int var4 = input[ dataW * (y+1) + x   ] - input[ dataW * (y-1) + x   ]; 
	
    const int magnitude = sqrt((float) ((var1+var2+2*var3)*(var1+var2+2*var3) + (var1-var2-2*var4)*(var1-var2-2*var4)));
	
	output[location] = ( (magnitude > threshold) ? 255 : 0); 		

}

__global__ void warmup(unsigned char *input, 
                       unsigned char *output){

	// Read Input Data
	/////////////////////////////////////////////////////////////////////////////

	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
	  
	int location = 	y*(gridDim.x*TILE_SIZE)+x;
			
	unsigned char value = 0;

    output[location] = value;

}

void gpu_function (unsigned char *data, 
                   unsigned int height, 
                   unsigned int width,
                   int threshold ){
    
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	
	// Both are the same size (CPU/GPU).
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
	
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
	
    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu, 
        data, 
        size*sizeof(char), 
        hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());

    // Execute algorithm

    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

	// Kernel Call
	#if defined(CUDA_TIMING)
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
        
        kernel<<<dimGrid, dimBlock>>>(input_gpu, 
                                      output_gpu,
                                      width,
                                      height,
                                      threshold);
                                      
        checkCuda(hipPeekAtLastError());                                     
        checkCuda(hipDeviceSynchronize());
	
	#if defined(CUDA_TIMING)
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));

    // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));

}

void gpu_warmup   (unsigned char *data, 
                   unsigned int height, 
                   unsigned int width){
    #if defined (WARMUP)                     
        int gridXSize = 1 + (( width - 1) / TILE_SIZE);
        int gridYSize = 1 + ((height - 1) / TILE_SIZE);
        
        int XSize = gridXSize*TILE_SIZE;
        int YSize = gridYSize*TILE_SIZE;
        
        // Both are the same size (CPU/GPU).
        int size = XSize*YSize;
        
        // Allocate arrays in GPU memory
        checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
        checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
        
        checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
                
        // Copy data to GPU
        checkCuda(hipMemcpy(input_gpu, 
            data, 
            size*sizeof(char), 
            hipMemcpyHostToDevice));

        checkCuda(hipDeviceSynchronize());
            
        // Execute algorithm
            
        dim3 dimGrid(gridXSize, gridYSize);
        dim3 dimBlock(TILE_SIZE, TILE_SIZE);
        
        warmup<<<dimGrid, dimBlock>>>(input_gpu, 
                                      output_gpu);
                                             
        checkCuda(hipDeviceSynchronize());
            
        // Retrieve results from the GPU
        checkCuda(hipMemcpy(data, 
                output_gpu, 
                size*sizeof(unsigned char), 
                hipMemcpyDeviceToHost));
                            
        // Free resources and end the program
        checkCuda(hipFree(output_gpu));
        checkCuda(hipFree(input_gpu));
    #endif
}

