#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "config.h"

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     
  
unsigned char *input_gpu;
unsigned char *output_gpu;

texture<unsigned char, 2, hipReadModeElementType> tex8u;

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}

__global__ void kernel(unsigned char *input, 
                       unsigned char *output,
                       unsigned int height, 
                       unsigned int width,
                       int threshold){

    const unsigned int x = blockIdx.x*TILE_SIZE+threadIdx.x;
    const unsigned int y = blockIdx.y*TILE_SIZE+threadIdx.y;
    
    // Skip this thread if outside the size
    if (x >= width-1 || y >= height-1)
        return;

    const unsigned int location= y*TILE_SIZE*gridDim.x+x;

    const char sobel_x[3][3] = {
        {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1}
    };

    const char sobel_y[3][3] = {
        {-1, -2, -1},
        { 0,  0,  0},
        { 1,  2,  1}
    };	

    unsigned int sum[2] ={0};

    #pragma unroll
    for (unsigned int i = 0 ; i < 3 ; ++i){
        #pragma unroll
        for (unsigned int j = 0 ; j < 3 ; ++j){
            const int x_elem = i+x-1;
            const int y_elem = j+y-1;
            sum[0] += tex2D(tex8u,x_elem,y_elem)*sobel_x[i][j];
            sum[1] += tex2D(tex8u,x_elem,y_elem)*sobel_y[i][j];
        }
    }

    const int magnitude = sqrt((float) ( sum[0]*sum[0] + sum[1]*sum[1]));

    if (magnitude > threshold)
        output[location] = 255;
    else
        output[location] = 0;		

}

__global__ void warmup(unsigned char *input, 
                       unsigned int height, 
                       unsigned int width,
                       unsigned char *output){

	// Read Input Data
	/////////////////////////////////////////////////////////////////////////////

	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
	  
    if (x >= width || y >= height) return;
    
	int location = 	y*(gridDim.x*TILE_SIZE)+x;
			
	unsigned char value = 0;

    output[location] = value;

}

void gpu_function (unsigned char *data, 
                   unsigned int height, 
                   unsigned int width,
                   int threshold ){
    
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	// Both are the same size (CPU/GPU).
	int size = height*width;//XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
	
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
	
    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu, 
        data, 
        size*sizeof(char), 
        hipMemcpyHostToDevice));

    // TEXTURE - Create texture parameters
    tex8u.addressMode[0] = hipAddressModeMirror; // 
    tex8u.addressMode[1] = hipAddressModeMirror; // Pad with zeros 
    tex8u.filterMode = hipFilterModePoint;
    tex8u.normalized = false;

    // TEXTURE - Bind Texture
    size_t pitch = sizeof(char)*width;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
    checkCuda(hipBindTexture2D(NULL, tex8u,
                input_gpu, channelDesc,
                width, height, pitch));

	checkCuda(hipDeviceSynchronize());

    // Execute algorithm

    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

	// Kernel Call
	#if defined(CUDA_TIMING)
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
        
        kernel<<<dimGrid, dimBlock>>>(input_gpu, 
                                      output_gpu,
                                      height,
                                      width,
                                      threshold);
                                      
        checkCuda(hipPeekAtLastError());                                     
        checkCuda(hipDeviceSynchronize());
	
	#if defined(CUDA_TIMING)
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));

    // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));

}

void gpu_warmup   (unsigned char *data, 
                   unsigned int height, 
                   unsigned int width){
    #if defined (WARMUP)                     
        int gridXSize = 1 + (( width - 1) / TILE_SIZE);
        int gridYSize = 1 + ((height - 1) / TILE_SIZE);
        
        // Both are the same size (CPU/GPU).
        int size = height*width;//XSize*YSize;
        
        // Allocate arrays in GPU memory
        checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
        checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
        
        checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
                
        // Copy data to GPU
        checkCuda(hipMemcpy(input_gpu, 
            data, 
            size*sizeof(char), 
            hipMemcpyHostToDevice));

        checkCuda(hipDeviceSynchronize());
            
        // Execute algorithm
            
        dim3 dimGrid(gridXSize, gridYSize);
        dim3 dimBlock(TILE_SIZE, TILE_SIZE);
        
        warmup<<<dimGrid, dimBlock>>>(input_gpu, 
                                      height,
                                      width,
                                      output_gpu);
                                             
        checkCuda(hipDeviceSynchronize());
            
        // Retrieve results from the GPU
        checkCuda(hipMemcpy(data, 
                output_gpu, 
                size*sizeof(unsigned char), 
                hipMemcpyDeviceToHost));
                            
        // Free resources and end the program
        checkCuda(hipFree(output_gpu));
        checkCuda(hipFree(input_gpu));
    #endif
}

