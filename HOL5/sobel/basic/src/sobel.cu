#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "config.h"

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     
  
unsigned char *input_gpu;
unsigned char *output_gpu;

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}

__global__ void kernel(unsigned char *input, 
                       unsigned char *output,
                       unsigned int size_x,
                       unsigned int size_y,
                       int threshold){

	// Read Input Data
	/////////////////////////////////////////////////////////////////////////////

	unsigned int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	unsigned int y = blockIdx.y*TILE_SIZE+threadIdx.y;
	  
	unsigned int location = 	y*(gridDim.x*TILE_SIZE)+x;

    // If thread out of image range, exit
    if (y >= 1 && y < size_y - 1 && x >= 1 && x < size_x - 1){
        	
        // Algorithm 
        /////////////////////////////////////////////////////////////////////

		int sum1 = input[ (gridDim.x*TILE_SIZE) * (y-1) + x+1 ] - 
                   input[ (gridDim.x*TILE_SIZE) * (y-1) + x-1 ] + 
               2 * input[ (gridDim.x*TILE_SIZE) * (y)   + x+1 ] - 
               2 * input[ (gridDim.x*TILE_SIZE) * (y)   + x-1 ] + 
                   input[ (gridDim.x*TILE_SIZE) * (y+1) + x+1 ] - 
                   input[ (gridDim.x*TILE_SIZE) * (y+1) + x-1 ];

		int sum2 = input[ (gridDim.x*TILE_SIZE) * (y-1) + x-1 ] + 
               2 * input[ (gridDim.x*TILE_SIZE) * (y-1) + x   ] + 
                   input[ (gridDim.x*TILE_SIZE) * (y-1) + x+1 ] - 
                   input[ (gridDim.x*TILE_SIZE) * (y+1) + x-1 ] - 
               2 * input[ (gridDim.x*TILE_SIZE) * (y+1) + x   ] - 
                   input[ (gridDim.x*TILE_SIZE) * (y+1) + x+1 ];

		int magnitude =  sqrt( (float) (sum1*sum1 + sum2*sum2));

		if (magnitude > threshold)
			output[location] = 255;
		else
			output[location] = 0;
	}

}

__global__ void warmup(unsigned char *input, 
                       unsigned char *output){

	// Read Input Data
	/////////////////////////////////////////////////////////////////////////////

	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
	  
	int location = 	y*(gridDim.x*TILE_SIZE)+x;
			
	unsigned char value = 0;

    output[location] = value;

}

void gpu_function (unsigned char *data, 
                   unsigned int height, 
                   unsigned int width,
                   int threshold ){
    
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	
	// Both are the same size (CPU/GPU).
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
	
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
	
    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu, 
        data, 
        size*sizeof(char), 
        hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());

    // Execute algorithm

    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

	// Kernel Call
	#if defined(CUDA_TIMING)
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
        
        kernel<<<dimGrid, dimBlock>>>(input_gpu, 
                                      output_gpu,
                                      width,
                                      height,
                                      threshold);
                                      
        checkCuda(hipPeekAtLastError());                                     
        checkCuda(hipDeviceSynchronize());
	
	#if defined(CUDA_TIMING)
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));

    // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));

}

void gpu_warmup   (unsigned char *data, 
                   unsigned int height, 
                   unsigned int width){
    #if defined (WARMUP)                     
        int gridXSize = 1 + (( width - 1) / TILE_SIZE);
        int gridYSize = 1 + ((height - 1) / TILE_SIZE);
        
        int XSize = gridXSize*TILE_SIZE;
        int YSize = gridYSize*TILE_SIZE;
        
        // Both are the same size (CPU/GPU).
        int size = XSize*YSize;
        
        // Allocate arrays in GPU memory
        checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
        checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
        
        checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
                
        // Copy data to GPU
        checkCuda(hipMemcpy(input_gpu, 
            data, 
            size*sizeof(char), 
            hipMemcpyHostToDevice));

        checkCuda(hipDeviceSynchronize());
            
        // Execute algorithm
            
        dim3 dimGrid(gridXSize, gridYSize);
        dim3 dimBlock(TILE_SIZE, TILE_SIZE);
        
        warmup<<<dimGrid, dimBlock>>>(input_gpu, 
                                      output_gpu);
                                             
        checkCuda(hipDeviceSynchronize());
            
        // Retrieve results from the GPU
        checkCuda(hipMemcpy(data, 
                output_gpu, 
                size*sizeof(unsigned char), 
                hipMemcpyDeviceToHost));
                            
        // Free resources and end the program
        checkCuda(hipFree(output_gpu));
        checkCuda(hipFree(input_gpu));
    #endif
}

