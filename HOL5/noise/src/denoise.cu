#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "config.h"

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     
  
unsigned char *input_gpu;
unsigned char *output_gpu;

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}

__device__ void insertion_sort(unsigned char A[], int size) {
    for (int ii = 0; ii < (size - 1); ++ii) {
        int min_idx = ii + 1;
        for (int jj = (ii + 1); jj < size; ++jj) {
            if (A[jj] < A[min_idx]) {
               min_idx = jj;
            } 
        }
        if (A[ii] > A[min_idx]) {
            int temp = A[ii];
            A[ii] = A[min_idx];
            A[min_idx] = temp;
        }
    }
}

__global__ void kernel(unsigned char *input, 
                       unsigned char *output,
                       unsigned int size_x,
                       unsigned int size_y,
                       int inc){

	// Read Input Data
	/////////////////////////////////////////////////////////////////////////////

	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
	  
	unsigned int location = y*(gridDim.x*TILE_SIZE)+x;
    
	 // If thread out of image range, exit
    if (y >= 1 && y < size_y - 1 && x >= 1 && x < size_x - 1){
        
        // Read neighbor values and store them in the array
        unsigned char values[9];

        // Write code here that can populate the values array with the neighboring pixel values
        int idx = 0;
        for (int ii = x-1; ii <= x+1; ++ii) {
            for (int jj = y-1; jj <= y+1; ++jj) {
                values[idx] = input[(jj*(gridDim.x*TILE_SIZE))+ii];
                ++idx;
            }
        }
        
        // Apply Bubble sort
        insertion_sort (values, 9);
        
        // Output is the value in the middle of the sorted array
        output[location] = values[4];
    }
    

}

__global__ void warmup(unsigned char *input, 
                       unsigned char *output){

	// Read Input Data
	/////////////////////////////////////////////////////////////////////////////

	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
	  
	int location = 	y*(gridDim.x*TILE_SIZE)+x;
			
	unsigned char value = 0;

    output[location] = value;

}

void gpu_function (unsigned char *data, 
                   unsigned int height, 
                   unsigned int width,
                   int inc ){
    
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	
	// Both are the same size (CPU/GPU).
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
	
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
	
    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu, 
        data, 
        size*sizeof(char), 
        hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());

    // Execute algorithm

    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

	// Kernel Call
	#if defined(CUDA_TIMING)
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
        
        kernel<<<dimGrid, dimBlock>>>(input_gpu, 
                                      output_gpu,
                                      width,
                                      height,
                                      inc);
                                      
        checkCuda(hipPeekAtLastError());                                     
        checkCuda(hipDeviceSynchronize());
	
	#if defined(CUDA_TIMING)
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));

    // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));

}

void gpu_warmup   (unsigned char *data, 
                   unsigned int height, 
                   unsigned int width){
    #if defined (WARMUP)                     
        int gridXSize = 1 + (( width - 1) / TILE_SIZE);
        int gridYSize = 1 + ((height - 1) / TILE_SIZE);
        
        int XSize = gridXSize*TILE_SIZE;
        int YSize = gridYSize*TILE_SIZE;
        
        // Both are the same size (CPU/GPU).
        int size = XSize*YSize;
        
        // Allocate arrays in GPU memory
        checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
        checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
        
        checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
                
        // Copy data to GPU
        checkCuda(hipMemcpy(input_gpu, 
            data, 
            size*sizeof(char), 
            hipMemcpyHostToDevice));

        checkCuda(hipDeviceSynchronize());
            
        // Execute algorithm
            
        dim3 dimGrid(gridXSize, gridYSize);
        dim3 dimBlock(TILE_SIZE, TILE_SIZE);
        
        warmup<<<dimGrid, dimBlock>>>(input_gpu, 
                                      output_gpu);
                                             
        checkCuda(hipDeviceSynchronize());
            
        // Retrieve results from the GPU
        checkCuda(hipMemcpy(data, 
                output_gpu, 
                size*sizeof(unsigned char), 
                hipMemcpyDeviceToHost));
                            
        // Free resources and end the program
        checkCuda(hipFree(output_gpu));
        checkCuda(hipFree(input_gpu));
    #endif
}

